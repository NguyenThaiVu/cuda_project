
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
#include <stdio.h>

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(1);\
    }\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};


void printDeviceInfo() 
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}


__global__ void AddTwoVectors(float A[], float B[], float C[], int N) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    C[i] = A[i] + B[i];
}

int main(void) 
{
    int N = 10000;
    float A[N], B[N], C[N]; // Arrays for vectors A, B, and C

    for (int i = 0; i < N; ++i) {
        A[i] = i;
        B[i] = i+1;
    }

    // Device pointers for vectors A, B, and C
    float *d_A, *d_B, *d_C; 

    hipMalloc((void **)&d_A, N * sizeof(float));
    hipMalloc((void **)&d_B, N * sizeof(float));
    hipMalloc((void **)&d_C, N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);    

    int BLOCK_SIZE = 256;
    int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    GpuTimer timer;
    timer.Start();
    AddTwoVectors<<<GRID_SIZE, BLOCK_SIZE>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());

    // Copy data from device to host
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    
    // for (int i = 0; i < N; ++i) {
    //     printf("C[%d] = %f\n", i, C[i]);
    // }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
