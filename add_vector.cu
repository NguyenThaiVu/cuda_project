
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void AddTwoVectors(float A[], float B[], float C[], int N) 
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main(void) 
{
    int N = 1000;
    float A[N], B[N], C[N]; // Arrays for vectors A, B, and C

    for (int i = 0; i < N; ++i) {
        A[i] = i;
        B[i] = i+1;
    }

    // Device pointers for vectors A, B, and C
    float *d_A, *d_B, *d_C; 

    hipMalloc((void **)&d_A, N * sizeof(float));
    hipMalloc((void **)&d_B, N * sizeof(float));
    hipMalloc((void **)&d_C, N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);    

    AddTwoVectors<<<1, N>>>(d_A, d_B, d_C, N);

    // Copy data from device to host
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    
    // Waits untill all CUDA threads are executed
    hipDeviceSynchronize();
    
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    for (int i = 0; i < N; ++i) {
        printf("C[%d] = %f\n", i, C[i]);
    }

    return 0;
}
