#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
#include <stdio.h>
#include </home/thaiv7/Desktop/cuda_project/utils/in_out_helper.h>
#include </home/thaiv7/Desktop/cuda_project/utils/matrix_utils.cu>
#include </home/thaiv7/Desktop/cuda_project/utils/gpu_helper.cu>

#define BLOCK_SIZE 32

inline double gflops_from_ms(long long M, long long N, long long K,
                             double elapsed_ms, int repeats = 1)
{
    // Total floating-point operation for GEMM
    long double ops = 2.0L * (long double)M * (long double)N * (long double)K * (long double)repeats;
    long double seconds = elapsed_ms / 1000.0L;
    long double gflops = ops / (seconds * 1.0e9L);
    return static_cast<double>(gflops);
}

__global__ void matmulDeviceKernel(Matrix d_A, Matrix d_B, Matrix d_C)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((row < d_C.height) && (col < d_C.width))
    {
        float value = 0;

        for (int k = 0; k < d_A.width; k++)
        {
            value += getElementMatrix(d_A, row, k) * getElementMatrix(d_B, k, col);
        }
        setElementMatrix(d_C, row, col, value);
    }
}

// TODO: implement for matrix size not multiple of BLOCK_SIZE
__global__ void matmulDeviceKernelSharedMem(Matrix d_A, Matrix d_B, Matrix d_C)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float subMatrixA_share[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float subMatrixB_share[BLOCK_SIZE][BLOCK_SIZE];

    int numTile = (d_A.width + BLOCK_SIZE - 1) / BLOCK_SIZE;
    float value = 0.0;
    for (int idxTile = 0; idxTile < numTile; idxTile++)
    {
        int sx = threadIdx.x;
        int sy = threadIdx.y;

        if ((row < d_A.height) && (idxTile * BLOCK_SIZE + sx < d_A.width))
            subMatrixA_share[sy][sx] = getElementMatrix(d_A, row, idxTile * BLOCK_SIZE + sx);
        else
            subMatrixA_share[sy][sx] = 0.0;

        if ((idxTile * BLOCK_SIZE + sy < d_B.height) && (col < d_B.width))
            subMatrixB_share[sy][sx] = getElementMatrix(d_B, idxTile * BLOCK_SIZE + sy, col);
        else
            subMatrixB_share[sy][sx] = 0.0;

        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; k++)
        {
            value += subMatrixA_share[sy][k] * subMatrixB_share[k][sx];
        }
        __syncthreads();
    }
    if ((row < d_C.height) && (col < d_C.width))
    {
        setElementMatrix(d_C, row, col, value);
    }
}

void matmulDevice(Matrix A, Matrix B, Matrix C)
{
    Matrix d_A, d_B, d_C;

    d_A.height = A.height;
    d_A.width = A.width;
    d_A.stride = A.stride;
    hipMalloc((void **)&d_A.arr, d_A.height * d_A.width * sizeof(float));
    hipMemcpy(d_A.arr, A.arr, d_A.height * d_A.width * sizeof(float), hipMemcpyHostToDevice);

    d_B.height = B.height;
    d_B.width = B.width;
    d_B.stride = B.stride;
    hipMalloc((void **)&d_B.arr, d_B.height * d_B.width * sizeof(float));
    hipMemcpy(d_B.arr, B.arr, d_B.height * d_B.width * sizeof(float), hipMemcpyHostToDevice);

    d_C.height = d_A.height;
    d_C.width = d_B.width;
    d_C.stride = C.stride;
    hipMalloc((void **)&d_C.arr, d_C.height * d_C.width * sizeof(float));

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((d_C.width + BLOCK_SIZE - 1) / BLOCK_SIZE, (d_C.height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Warm up
    // matmulDeviceKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    for (int i = 0; i < 10; ++i)
    {
        matmulDeviceKernelSharedMem<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
        hipDeviceSynchronize();
    }

    GpuTimer timer;
    float totalTime = 0;

    for (int i = 0; i < 10; ++i)
    {
        timer.Start();
        // matmulDeviceKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
        matmulDeviceKernelSharedMem<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
        hipDeviceSynchronize(); // wait to device computation is finished.
        timer.Stop();
        float ms = timer.Elapsed();
        totalTime += ms;
    }
    float ms = totalTime / 10.0f;
    printf("Kernel execution time: %f ms\n", ms);

    double gflops = gflops_from_ms(d_A.height, B.width, B.height, ms);
    printf("Performance: %f GFLOPS\n", gflops);

    hipMemcpy(C.arr, d_C.arr, d_C.height * d_C.width * sizeof(float), hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipFree(d_A.arr);
    hipFree(d_B.arr);
    hipFree(d_C.arr);
}

int main()
{
    int m = 512;
    int n = 512;
    int p = 512;

    Matrix A, B, C, C_host;
    initMatrixHost(&A, m, p, 1.0);
    initMatrixHost(&B, p, n, 1.0);
    initMatrixHost(&C_host, m, n, 0.0);
    initMatrixHost(&C, m, n, 0.0);

    matmulHost(A, B, C_host);
    matmulDevice(A, B, C);

    bool isSimilar = compare2Matrix(C, C_host);
    if (isSimilar == true)
    {
        cout << "CUDA implementation is correct" << endl;
    }
    else
    {
        cout << "CUDA is IN-CORRECT !" << endl;
    }

    // printMatrix(A);
    // printMatrix(B);
    // printMatrix(C_host);
    // printMatrix(C);

    free(A.arr);
    free(B.arr);
    free(C_host.arr);
    free(C.arr);
    return 0;
}