
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
#include <stdio.h>
#include <fstream>
#include <time.h>

/*
This file define helper functions for CUDA programming, such as: error checking, print device info, save data to file, etc.
*/

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(1);\
    }\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};


void printDeviceInfo() 
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Max threads per block: %d\n", devProv.maxThreadsPerBlock);
    printf("Max threads dimensions (x,y,z): (%d, %d, %d)\n",
           devProv.maxThreadsDim[0], devProv.maxThreadsDim[1], devProv.maxThreadsDim[2]);
    printf("Max grid size (x,y,z): (%d, %d, %d)\n",
           devProv.maxGridSize[0], devProv.maxGridSize[1], devProv.maxGridSize[2]);

    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("Max block per SM: %d\n", devProv.maxBlocksPerMultiProcessor);
    
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}


__device__ float relu(float x) 
{
    return x > 0.0f ? x : 0.0f;
}

__global__ void applyReLU_vector(float* data, int size) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        data[tid] = relu(data[tid]);
    }
}